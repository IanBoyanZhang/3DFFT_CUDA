// Multi-Dimensional cufft test
// Complex to Complex
// Un-normalized
#include <math.h>
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
//#include <cutil_inline.h>

/* Define a type mult-dim data input to be a struct with integer members x,y,z */
typedef struct {
	int x;
	int y;
	int z;
} data_size;
// declaration forward
void runTest(hipfftComplex *, data_size, int, int, int *);
static __device__ __host__ inline hipfftComplex ComplexScale(hipfftComplex, float);
// Program main
int main(int argc, char *argv[]){
	
	data_size data_sz;
		
	/* from user cmd line input */
	int nx = 128;
	int ny = 128;
	int nz = 128;
	int batch = 1;
	// Transform Dimension
	const int nrank = 3;
	/* Check dimension match or not */
	if ((nz>1 && nrank<3) || (ny>1 && nrank<2)){
		fprintf(stderr, "Input data and transform dimension do not match, quit\n");
		return 1;
	}		

	data_sz.x = nx;
	data_sz.y = ny;
	data_sz.z = nz;	

	//int n[nrank] = {nx, ny};
	int n[nrank] = {nx, ny, nz};
	/* Host memory allocation */	
	hipfftComplex* data = (hipfftComplex*)malloc(sizeof(hipfftComplex)*nx*ny*nz*batch);
	/* source data creation */
	for (unsigned int i=0; i<nx*ny*nz; i++){
		data[i].x = 1.0f;
		data[i].y = 1.0f;
	}
	
	runTest(data, data_sz, nrank, batch, n);
	free(data);	
	// printf("Seg\n");

	/* for (int i=0; i<NX*NY*BATCH; i++){
		printf("data[%d] %f %f\n", i, data[i].x, data[i].y);
	}
	*/
	return 0;
}

// ! Run a simple test with Transform size --
//void runTest2d(hipfftComplex* dataPtr, int NX, int NY, int NRANK, int BATCH, int * n){
void runTest(hipfftComplex* dataPtr, data_size data_sz, int NRANK, int BATCH, int * n){
	
	hipfftHandle plan;
	hipfftComplex *devPtr;
	int NX = data_sz.x;
	int NY = data_sz.y;
	int NZ = data_sz.z;
	/* GPU memory allocation */
	//hipMalloc((void**)&devPtr, sizeof(hipfftComplex)*NX*NY*NZ*BATCH);
	hipMalloc((void**)&devPtr, sizeof(hipfftComplex)*NX*NY*NZ*BATCH);

	/* Error Checker */
	if (hipGetLastError()!= hipSuccess){
		fprintf(stderr, "Cuda error: Failed to allocate\n");
		// return 1;
	}

	/* transfer to GPU memory */
	hipMemcpy(devPtr, dataPtr, sizeof(hipfftComplex)*NX*NY*NZ*BATCH, hipMemcpyHostToDevice);
	if (hipGetLastError()!= hipSuccess){
		fprintf(stderr, "hipMemcpy failed!\n");
		// return 1;
	}
	
	/* One device memory */
//	hipMemset(data, 1, sizeof());

#ifdef CUDA_TIMER
	hipEvent_t start_event, stop_event;
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);
#endif

#ifdef CUDA_TIMER
	hipEventRecord(start_event, 0);
	float t_device;
#else
	hipDeviceSynchronize();
	double t_device = -getTime();
#endif

	/* Create a mult-D FFT plan */
	if (hipfftPlanMany(&plan, NRANK, n, NULL, 1, NX*NY*NZ, // *inembed, istride, idist
					NULL, 1, NX*NY*NZ, // *onembed, ostride, odist
					HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: Plan creation failed\n");
		//return 1;
	}

	/* Use the CUFFT plan to transform the signal in place */
	/* executes Forward  FFT */
	if (hipfftExecC2C(plan, devPtr, devPtr, HIPFFT_FORWARD) != HIPFFT_SUCCESS)		{
		fprintf(stderr, "Cuda error: ExecC2C Forward failed\n");
		// return 1;
	}

	if (hipDeviceSynchronize() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to synchronize\n");
		// return 1;
	}
#ifdef CUDA_TIMER
	hipEventRecord(stop_event, 0);
	hipEventSynchronize(stop_event);
	hipEventElapsedTime(&t_device, start_event, stop_event);

#else
	/* block until the device has finished */
	hipDeviceSynchronize();
	/* stop the timer */
	t_device += getTime();
#endif
	fprintf(stdout, "Time: %f\n", t_device);
	
	/* executes Inverse FFT */
	hipfftExecC2C(plan, devPtr, devPtr, HIPFFT_BACKWARD);
	/* transfer results from GPU memory */
	hipMemcpy(dataPtr, devPtr, sizeof(hipfftComplex)*NX*NY*NZ*BATCH, hipMemcpyDeviceToHost);
	/* deletes CUFFT plan */
	hipfftDestroy(plan);
	/* frees GPU memory */
	hipFree(devPtr);
	// free(data);
	hipDeviceReset();
	// return 0;
}
