#include "hip/hip_runtime.h"
// 1D fft vector test
// Complex to Complex
// Normalized
// Forward
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

//#include "types.h"
//#include "utils.h"
#include <sys/time.h>
#include <stdio.h>

#define NX	128
#define NY	128
//#define BATCH	10
#define BATCH	128
//#define NRANK	3
#define NRANK	1

void genMatrix( float *a, unsigned int m, unsigned int n);
void verify( float *C, unsigned int m, unsigned int n, float eps, char *mesg);
void verify( float *c_d, float *c_h, unsigned int m, unsigned int n, float eps, char *mesg);
void printMatrix( float *a, unsigned int m, unsigned int n);
double getTime();
//double gflops(int n, int niter, double time);

// declaration forward
void runTest1d(hipfftComplex *, int);
static __device__ __host__ inline hipfftComplex ComplexScale(hipfftComplex, float);
static __global__ void ComplexPointwiseScale(hipfftComplex*, int, float);
// Program main

int main(int argc, char *argv[]){

	//int n[NRANK] = {NX, NY, NZ};
	/* Host memory allocation */	
	hipfftComplex* h_data = (hipfftComplex*)malloc(sizeof(hipfftComplex)*NX*NY*BATCH);
	const int size = NX * NY* BATCH;
	/* source data creation */
	for (unsigned int i=0; i<NX*NY*BATCH; i++){
		h_data[i].x = 1.0f;
		h_data[i].y = 1.0f;
	}
	
	runTest1d(h_data, size);
	free(h_data);	

	return 0;
}

// ! Run a simple test with Transform size --
void runTest1d(hipfftComplex* dataPtr, int size){
	
	const int SCALE = 10;

	hipfftHandle plan;
	hipfftComplex *devPtr;

	hipfftComplex *dummy_devPtr;
	
	/* GPU memory allocation */
	hipMalloc((void**)&devPtr, sizeof(hipfftComplex)*NX*NY*BATCH);
	hipMalloc((void**)&dummy_devPtr, sizeof(hipfftComplex)*NX*NY*BATCH);
	/* Error Checker */
	if (hipGetLastError()!= hipSuccess){
		fprintf(stderr, "Cuda error: Failed to allocate\n");
		// return 1;
	}

	/* transfer to GPU memory */
	hipMemcpy(devPtr, dataPtr, sizeof(hipfftComplex)*NX*NY*BATCH, hipMemcpyHostToDevice);
	/* cpy data to dummy ptr memory */	
	hipMemcpy(dummy_devPtr, dataPtr, sizeof(hipfftComplex)*NX*NY*BATCH, hipMemcpyHostToDevice);
	if (hipGetLastError()!= hipSuccess){
		fprintf(stderr, "hipMemcpy failed!\n");
		// return 1;
	}
	/* One device memory */
//	hipMemset(data, 1, sizeof())
	// Start the timer
#ifdef CUDA_TIMER
	hipEvent_t start_event, stop_event;
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);
#endif


	/* Warm up */
	/* execute cufft forward transform */
	/* Create a 1D FFT plan */
	if (hipfftPlan1d(&plan, NX, HIPFFT_C2C, NY*BATCH) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: Plan creation failed");
		//return 1;
	}

	/* executes Forward  FFT */
	/* Identical pointers to input and output arrays implies in-place transformation*/
	if (hipfftExecC2C(plan, devPtr, devPtr, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stderr, "Cuda error: ExecC2C Forward failed\n");
		// return 1;
	}
	if (hipDeviceSynchronize() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to synchronize\n");
		// return 1;
	}
#ifdef CUDA_TIMER
	hipEventRecord(start_event, 0);
	float t_device;
#else
	hipDeviceSynchronize();
	double t_device = -getTime();
#endif
	/* multi-run for timing */
	for(int r=0; r<SCALE; r++)
	{
		/* Create a 1D FFT plan */
		if (hipfftPlan1d(&plan, NX, HIPFFT_C2C, NY*BATCH) != HIPFFT_SUCCESS){
			fprintf(stderr, "CUFFT error: Plan creation failed");
			//return 1;
		}

		/* executes Forward  FFT */
		/* Identical pointers to input and output arrays implies in-place transformation*/
		if (hipfftExecC2C(plan, dummy_devPtr, dummy_devPtr, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
			fprintf(stderr, "Cuda error: ExecC2C Forward failed\n");
			// return 1;
		}	
		if (hipDeviceSynchronize() != hipSuccess){
			fprintf(stderr, "Cuda error: Failed to synchronize\n");
			// return 1;
		}
	}

#ifdef CUDA_TIMER
	hipEventRecord(stop_event, 0);
	hipEventSynchronize(stop_event);
	/* hipEventElapsedTime returns value in milliseconds. Resolution ~0.5ms */
	hipEventElapsedTime(&t_device, start_event, stop_event);
	t_device /= SCALE;
	//t_device /= 1000.0;

#else
	/* block until the device has finished */
	hipDeviceSynchronize();
	/* stop the timer */
	t_device += getTime();
#endif

	/* Normalizing Scale down */
	/* 1D Thread Structure */
	dim3 blocks(size / 128);
	/* host calling */
//	ComplexPointwiseScale<<<blocks, 128>>>(devPtr, size, 1.0f/128);
	/* executes Inverse FFT */
//	hipfftExecC2C(plan, devPtr, devPtr, HIPFFT_BACKWARD);

	/* transfer results from GPU memory */
	hipMemcpy(dataPtr, devPtr, sizeof(hipfftComplex)*NX*NY*BATCH, hipMemcpyDeviceToHost);

	/* deletes CUFFT plan */
	hipfftDestroy(plan);

	/* frees GPU memory */
	hipFree(devPtr);
	hipFree(dummy_devPtr);
	for (int i = 0; i < NX; i++ )
	{
		printf("dataPtr[%d] = %f %f\n", i, dataPtr[i].x, dataPtr[i].y);
	}
	// free(data);

	printf("Time: %f\n", t_device);	

	//double gflops_d = gflops(size, SCALE, t_device);
	//printf("Device computation time: %f sec. [%f gflops]\n", t_device, gflops_d);
	
	hipDeviceReset();

	// return 0;
}

static __device__ __host__ inline hipfftComplex ComplexScale(hipfftComplex a, float s)
{
	hipfftComplex c;
	c.x = s * a.x;
	c.y = s * a.y;
	return c;
}

static __global__ void ComplexPointwiseScale(hipfftComplex* a, int size, float scale)
{
	/* 128 to 128 one to one mapping */
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = threadID; i < size; i += numThreads)
		a[i] = ComplexScale(a[i], scale);
}

double getTime()
{
	const double kMicro = 1.0e-6;
	struct timeval TV;

	const int RC = gettimeofday(&TV, NULL);
	if(RC == -1)
	{
		printf("ERROR: Bad call to gettimeofday\n");
		return(-1);
	}

	return( ((double)TV.tv_sec) + kMicro * ((double)TV.tv_usec) );

}
/*
double gflops(int n, int reps, double time){

    // Total number of entries
    long long int n2 = n;
    n2 *= n;
    // Updates
    const long long int updates =  n2 * (long long) reps;
    // Number of flops
    const long long int flops =  (long long ) n * 2L * updates;
    double flop_rate = (double) flops / time;
    return ( flop_rate/1.0e9);
}
*/
